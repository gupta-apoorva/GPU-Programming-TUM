#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}


int main(int argc, char **argv)
{
	int n = 1000000;    
	float* a = new float[n];
	for (int i=0 ; i<n; i++){
	a[i] = i;
	}

	float* a_cuda;
	hipMalloc((void**)&a_cuda, n*sizeof(float));
	hipMemcpy(a_cuda, a , n*sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float sum_l =0;
	hipEventRecord(start);
	    hipblasSasum(handle, n, a_cuda, 1, &sum_l);
	hipEventRecord(stop);

	    
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "ARRAY SUM: " << sum_l  << " Time: "<< milliseconds << endl;


	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipblasDestroy(handle);
	free(a);
	hipFree(a_cuda);

	return 0;
}



