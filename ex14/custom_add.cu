#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}


__global__ void summing(float* a_cuda, float* b_cuda, int n){
    int t_numx = threadIdx.x + blockDim.x*blockIdx.x;
    int tid = threadIdx.x;

    __shared__ float blocksum;
if (tid ==0){
blocksum = 0;}

__syncthreads();

if (t_numx <n){
    atomicAdd(&blocksum, a_cuda[t_numx]);
}
__syncthreads();
if (tid ==0){    
    b_cuda[blockIdx.x] += blocksum;
}
}


int main(int argc, char **argv)
{
	int n = 1000000;    
	float* a = new float[n];
	for (int i=0 ; i<n; i++){
		a[i] = i;
	}

	float* a_cuda, *b_cuda;
	hipMalloc((void**)&a_cuda, n*sizeof(float));
	hipMemcpy(a_cuda, a , n*sizeof(float), hipMemcpyHostToDevice);

	int block = 1024;
	int grid = ((block + n -1)/block);

	hipMalloc((void**)&b_cuda,  grid*sizeof(float));
	hipMemset(b_cuda, 0 , grid*sizeof(float));   

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* b = new float[grid]; 
	float sum_l =0;
	hipEventRecord(start);

	summing <<<grid, block>>> (a_cuda, b_cuda , n);

	hipMemcpy(b , b_cuda, grid*sizeof(float) , hipMemcpyDeviceToHost);
	for (int i=0 ; i<grid ; i++){
		sum_l +=b[i];
	}

	hipEventRecord(stop);

    
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "ARRAY SUM: " << sum_l  << " Time: "<< milliseconds << endl;
	

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(b);
	hipFree(a_cuda);
	hipFree(b_cuda);

return 0;
}



