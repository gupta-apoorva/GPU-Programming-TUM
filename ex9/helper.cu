#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "helper.h"
#include <cstdlib>
#include <iostream>
using std::stringstream;
using std::cerr;
using std::cout;
using std::endl;
using std::string;




// parameter processing: template specialization for T=bool
template<>
bool getParam<bool>(std::string param, bool &var, int argc, char **argv)
{
    const char *c_param = param.c_str();
    for(int i=argc-1; i>=1; i--)
    {
        if (argv[i][0]!='-') continue;
        if (strcmp(argv[i]+1, c_param)==0)
        {
            if (!(i+1<argc) || argv[i+1][0]=='-') { var = true; return true; }
            std::stringstream ss;
            ss << argv[i+1];
            ss >> var;
            return (bool)ss;
        }
    }
    return false;
}




// opencv helpers
void convert_layered_to_interleaved(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[(nc-1-c) + nc*(x + (size_t)w*y)] = aIn[x + (size_t)w*y + nOmega*c];
            }
        }
    }
}
void convert_layered_to_mat(cv::Mat &mOut, const float *aIn)
{
    convert_layered_to_interleaved((float*)mOut.data, aIn, mOut.cols, mOut.rows, mOut.channels());
}


void convert_interleaved_to_layered(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[x + (size_t)w*y + nOmega*c] = aIn[(nc-1-c) + nc*(x + (size_t)w*y)];
            }
        }
    }
}
void convert_mat_to_layered(float *aOut, const cv::Mat &mIn)
{
    convert_interleaved_to_layered(aOut, (float*)mIn.data, mIn.cols, mIn.rows, mIn.channels());
}



void showImage(string title, const cv::Mat &mat, int x, int y)
{
    const char *wTitle = title.c_str();
    cv::namedWindow(wTitle, CV_WINDOW_AUTOSIZE);
    cvMoveWindow(wTitle, x, y);
    cv::imshow(wTitle, mat);
}

void showHistogram256(const char *windowTitle, int *histogram, int windowX, int windowY)
{
  const int nbins = 256;
  cv::Mat canvas = cv::Mat::ones(125, 512, CV_8UC3);

  float hmax = 0;
  for(int i = 0; i < nbins; ++i)
    hmax = max((int)hmax, histogram[i]);

  for (int j = 0, rows = canvas.rows; j < nbins-1; j++)
  {
    for(int i = 0; i < 2; ++i)
      cv::line(
        canvas, 
        cv::Point(j*2+i, rows), 
        cv::Point(j*2+i, rows - (histogram[j] * 125.0f) / hmax), 
        cv::Scalar(255,128,0), 
        1, 8, 0
        );
  }

  showImage(windowTitle, canvas, windowX, windowY);
}




// adding Gaussian noise
float noise(float sigma)
{
    float x1 = (float)rand()/RAND_MAX;
    float x2 = (float)rand()/RAND_MAX;
    return sigma * sqrtf(-2*log(std::max(x1,0.000001f)))*cosf(2*M_PI*x2);
}
void addNoise(cv::Mat &m, float sigma)
{
    float *data = (float*)m.data;
    int w = m.cols;
    int h = m.rows;
    int nc = m.channels();
    size_t n = (size_t)w*h*nc;
    for(size_t i=0; i<n; i++)
    {
        data[i] += noise(sigma);
    }
}




// cuda error checking
string prev_file = "";
int prev_line = 0;
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        exit(1);
    }
    prev_file = file;
    prev_line = line;
}
