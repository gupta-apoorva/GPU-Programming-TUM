// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__global__ void add(float* cuda_a, float* cuda_b, float* cuda_c, int n){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;
    if (t_numx + t_numy + t_numz < n){
        cuda_c[t_numx + t_numy + t_numz] = cuda_a[t_numx + t_numy + t_numz] + cuda_b[t_numx + t_numy + t_numz];
    }
}


int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 20;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    for(int i=0; i<n; i++)
    {
        a[i] = i;
        b[i] = (i%5)+1;
        c[i] = 0;
    }

    // CPU computation
    for(int i=0; i<n; i++) c[i] = a[i] + b[i];

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;
    // init c
    for(int i=0; i<n; i++) c[i] = 0;
    

    float *cuda_a, *cuda_b, *cuda_c;
    hipMalloc((void**)&cuda_a, n*sizeof(float));
    hipMalloc((void**)&cuda_b, n*sizeof(float));
    hipMalloc((void**)&cuda_c, n*sizeof(float));
    hipMemcpy(cuda_a, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_c, c, n*sizeof(float), hipMemcpyHostToDevice);

    dim3 block = dim3(128,1,1);
    int grid_x = ((n + block.x + 1)/block.x);
    int grid_y = 1;
    int grid_z = 1;
    dim3 grid = dim3(grid_x, grid_y, grid_z );

    add <<<block,grid>>> (cuda_a,cuda_b, cuda_c, n);

    hipMemcpy(c, cuda_c, n*sizeof(int), hipMemcpyDeviceToHost);


    
    // print result
    cout << "GPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    return 0;
}



