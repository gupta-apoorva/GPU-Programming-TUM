#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;


// uncomment to use the camera
//#define CAMERA

__device__ void eigenvalues (float m11,float m12,float m22, float* lambda1,float* lambda2){
    float T = m11 + m22;
    float D =  m11*m22 - m12*m12;

    *lambda1 = T/2.f + sqrt(pow(T,2.f)/4.f - D);
    *lambda2 = T/2.f - sqrt(pow(T,2.f)/4.f - D);
}

__global__ void T_eigenvalues(float* imgIn_cuda, float* imgOut_cuda,float* T_out_cuda, int  w, int h, int nc, float alpha, float beta){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = 0;

    float lambda1, lambda2;
    float m11, m12, m22;
    if (t_numx < w && t_numy < h && t_numz < nc){
        m11 = T_out_cuda[t_numx + w*t_numy + w*h*0];
        m12 = T_out_cuda[t_numx + w*t_numy + w*h*1];
        m22 = T_out_cuda[t_numx + w*t_numy + w*h*2];

        //float T = m11 + m22;
        //float D =  m11*m22 - m12*m12;

        //lambda1 = T/2.f + sqrt(pow(T,2.f)/4.f - D);
        //lambda2 = T/2.f - sqrt(pow(T,2.f)/4.f - D);
        eigenvalues(m11, m12, m22, &lambda1, &lambda2);

        if (lambda1 >= lambda2 && lambda2 >= alpha){
            imgOut_cuda[t_numx + w*t_numy + w*h*0] = 255;
            imgOut_cuda[t_numx + w*t_numy + w*h*1] = 0;
            imgOut_cuda[t_numx + w*t_numy + w*h*2] = 0;
        }
        else if (lambda1 >= alpha && alpha > beta && beta >= lambda2){
            imgOut_cuda[t_numx + w*t_numy + w*h*0] = 255;
            imgOut_cuda[t_numx + w*t_numy + w*h*1] = 255;
            imgOut_cuda[t_numx + w*t_numy + w*h*2] = 0;
        }
        else{
            imgOut_cuda[t_numx + w*t_numy + w*h*0] = 0.5*imgIn_cuda[t_numx + w*t_numy + w*h*0];
            imgOut_cuda[t_numx + w*t_numy + w*h*1] = 0.5*imgIn_cuda[t_numx + w*t_numy + w*h*1];
            imgOut_cuda[t_numx + w*t_numy + w*h*2] = 0.5*imgIn_cuda[t_numx + w*t_numy + w*h*2];
        }
    }
}


__global__ void gradient_fd(float* imgOut_cuda,float* cuda_v1,float* cuda_v2,int w,int h,int nc){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    if (t_numx + 1 < w && t_numy < h && t_numz < nc){
        cuda_v1[t_numx + w*t_numy + w*h*t_numz] = imgOut_cuda[t_numx + 1 + w*t_numy + w*h*t_numz] - imgOut_cuda[t_numx + w*t_numy + w*h*t_numz];
    }

    if (t_numx < w && t_numy + 1< h && t_numz < nc){
        cuda_v2[t_numx + w*t_numy + w*h*t_numz] = imgOut_cuda[t_numx + w*(t_numy+1) + w*h*t_numz] - imgOut_cuda[t_numx + w*t_numy + w*h*t_numz];
    }
}

__global__ void gradient_rand(float* imgOut_cuda,float* cuda_v1,float* cuda_v2,int w,int h,int nc){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    if (t_numx < w && t_numy < h && t_numz < nc){
        int x_neg = t_numx-1;
        int x_pos = t_numx+1;
        int y_neg = t_numy-1;
        int y_pos = t_numy+1;
        if (x_neg<0) x_neg = 0;
        if (x_pos>=w) x_pos = w-1;
        if (y_neg<0) y_neg = 0;
        if (y_pos>=h) y_pos = h-1;

        cuda_v1[t_numx + w*t_numy + w*h*t_numz] = 1.f/32.f*(3*imgOut_cuda[x_pos + w*y_pos + w*h*t_numz] 
                                        + 10*imgOut_cuda[x_pos + w*t_numy + w*h*t_numz] 
                                        + 3*imgOut_cuda[x_pos + w*y_neg + w*h*t_numz] 
                                        - 3*imgOut_cuda[x_neg + w*y_pos + w*h*t_numz]
                                        - 10*imgOut_cuda[x_neg + w*t_numy + w*h*t_numz] 
                                        - 3*imgOut_cuda[x_neg + w*y_neg + w*h*t_numz]) ;

        cuda_v2[t_numx + w*t_numy + w*h*t_numz] = 1.f/32.f*( 3*imgOut_cuda[x_pos + w*y_pos + w*h*t_numz] 
                                        + 10*imgOut_cuda[t_numx + w*y_pos + w*h*t_numz] 
                                        + 3*imgOut_cuda[x_neg + w*y_pos + w*h*t_numz] 
                                        - 3*imgOut_cuda[x_pos + w*y_neg + w*h*t_numz]
                                        - 10*imgOut_cuda[t_numx + w*y_neg + w*h*t_numz] 
                                        - 3*imgOut_cuda[x_neg + w*y_neg + w*h*t_numz]) ;
    }
}


__host__ void gaussian_kernel(float *kernel, float sigma, int radius){
    float sum = 0 ;
    for(int j = -radius; j<=radius ; j++){
            for (int i=-radius; i<=radius; i++){
                int index = i+radius + (2*radius+1)*(j+radius);
                kernel[index] = 0.5/3.14159/pow(sigma,2.0)*pow(2.71828,-(pow(i,2) + pow(j,2))/2/(pow(sigma,2)));  
                sum = sum +  kernel[index];   
        }
    }

    for (int i=0; i<(2*radius + 1)*(2*radius + 1); i++){
        kernel[i] = kernel[i] / sum;
    }
}

__global__ void cuda_convolution(float *imgIn_cuda, float* imgOut_cuda, float* kernel_cuda, int w, int h, int nc, int radius){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    if (t_numx < w && t_numy < h && t_numz < nc){
        int index = t_numx + w*t_numy + w*h*t_numz;
        //imgOut_cuda[index] = 0;
        for (int p = -radius; p <= radius; p++) {
            for (int q =-radius; q <= radius; q++) {
                int temp_j = t_numy + p;
                int temp_i = t_numx + q;
                if (temp_i<0) temp_i = 0;
                if (temp_i>=w) temp_i = w-1;
                if (temp_j<0) temp_j = 0;
                if (temp_j>=h) temp_j = h-1;
                int image_index = temp_i + temp_j*w + t_numz*h*w;
                int kernel_index = q+radius + (2*radius+1)*(p+radius);
                imgOut_cuda[index] += imgIn_cuda[image_index] * kernel_cuda[kernel_index];
            }
        }
    }    
}

__global__ void M_calculation(float *M_out_cuda, float* cuda_v1, float* cuda_v2, int w, int h, int nc){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    int index = t_numx + w*t_numy + w*h*t_numz;
    if (t_numx < w && t_numy < h && t_numz < nc){
        M_out_cuda[t_numx + w*t_numy + w*h*t_numz] += cuda_v1[index]*cuda_v1[index];
        M_out_cuda[t_numx + w*t_numy + w*h*t_numz] += cuda_v1[index]*cuda_v2[index];
        M_out_cuda[t_numx + w*t_numy + w*h*t_numz] += cuda_v2[index]*cuda_v2[index];
    }

}



int main(int argc, char **argv)
{
    hipDeviceSynchronize();  CUDA_CHECK;

#ifdef CAMERA
#else
    // input image
    string image = "";
    float sigma = 1;
    float alpha = 0.01;
    float beta = 0.001;
    bool ret = getParam("i", image, argc, argv);
    if (!ret) {cerr << "ERROR: no image specified" << endl; return 1;}
    bool ret2 = getParam("sigma", sigma, argc, argv);
    if (!ret2) {cerr << "ERROR: no sigma specified" << endl; return 1;}
    bool ret3 = getParam("alpha", alpha, argc, argv);
    if (!ret3) {cerr << "ERROR: no alpha specified" << endl; return 1;}
    bool ret4 = getParam("beta", beta, argc, argv);
    if (!ret4) {cerr << "ERROR: no beta specified" << endl; return 1;}
    if (argc <= 4) { cout << "Usage: " << argv[0] << " -i <image> -sigma <sigma> -alpha <alpha> -beta <beta>[-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
    cv::VideoCapture camera(0);
    if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
    camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
    camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;
    
    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

// CONVOLUTION ON GPU......START

    int radius = ceil(3*sigma);

    int total_ele_filter = (int)pow(2*radius + 1, 2);
    float* kernel = new float[total_ele_filter];

    gaussian_kernel(kernel, sigma, radius);


    float *kernel_cuda, *imgIn_cuda, *imgOut_cuda;
    hipMalloc((void**)&kernel_cuda, total_ele_filter*sizeof(float));
    hipMemcpy(kernel_cuda, kernel, total_ele_filter*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&imgIn_cuda , h*w*nc*sizeof(float));
    hipMalloc((void**)&imgOut_cuda , h*w*nc*sizeof(float));
    hipMemset(&imgOut , 0, h*w*nc*sizeof(float));
    hipMemcpy(imgIn_cuda, imgIn , h*w*nc*sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(imgOut_cuda, imgOut , h*w*nc*sizeof(float) , hipMemcpyHostToDevice);

    dim3 block = dim3(32,32,1);
    int grid_x = ((w + block.x - 1)/block.x);
    int grid_y = ((h + block.y - 1)/block.y);
    int grid_z = ((nc + block.z - 1)/block.z);
    dim3 grid = dim3(grid_x, grid_y, grid_z );

    cuda_convolution <<<grid, block>>> (imgIn_cuda, imgOut_cuda, kernel_cuda, w, h, nc, radius);
 
// CONVOLUTION ON GPU......END

// GRADIENT CALCULATION START
    int array_size = w*h*nc;

    float* cuda_v1;
    float* cuda_v2;

    hipMalloc((void**) &cuda_v1, array_size*sizeof(float));
    hipMalloc((void**) &cuda_v2, array_size*sizeof(float));

    hipMemset(cuda_v1, 0 , array_size*sizeof(float));
    hipMemset(cuda_v2, 0 , array_size*sizeof(float));


    gradient_rand <<<grid, block>>>(imgOut_cuda, cuda_v1, cuda_v2, w, h, nc );

    //hipMemcpy(imgOut, cuda_v2 , w*h*nc*sizeof(float) , hipMemcpyDeviceToHost);

// GRADIENT CALCULATION END

    float *M_out_cuda;

    hipMalloc((void**)&M_out_cuda , h*w*nc*sizeof(float));
    hipMemcpy(M_out_cuda, imgOut , h*w*nc*sizeof(float) , hipMemcpyHostToDevice);

    M_calculation <<< grid, block >>> (M_out_cuda, cuda_v1, cuda_v2, w, h, nc );

    float *T_out_cuda;

    hipMalloc((void**)&T_out_cuda , h*w*nc*sizeof(float));
    hipMemcpy(T_out_cuda, imgOut , h*w*nc*sizeof(float) , hipMemcpyHostToDevice);  

    cuda_convolution <<< grid, block >>> (M_out_cuda, T_out_cuda, kernel_cuda, w, h, nc, radius );

    T_eigenvalues <<< grid, block >>> ( imgIn_cuda, imgOut_cuda, T_out_cuda, w, h, nc, alpha, beta);

    float* T_out = new float [array_size];

    hipMemcpy(imgOut, imgOut_cuda , w*h*nc*sizeof(float) , hipMemcpyDeviceToHost);

    convert_layered_to_mat(mOut,imgOut);
    showImage("OUTPUT", mOut, 0, 0);
    showImage("Input", mIn, 100, 100); 
#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] kernel;
    delete[] T_out;

    hipFree(cuda_v1);
    hipFree(cuda_v2);
    hipFree(kernel_cuda);
    hipFree(imgOut_cuda);
    hipFree(imgIn_cuda);
    hipFree(T_out_cuda);
    hipFree(M_out_cuda);

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



