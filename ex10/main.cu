#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;


// uncomment to use the camera
//#define CAMERA

__host__ void gaussian_kernel(float *kernel, float sigma, int radius){
    float sum = 0 ;
    for(int j = -radius; j<=radius ; j++){
            for (int i=-radius; i<=radius; i++){
                    int index = i+radius + (2*radius+1)*(j+radius);
                kernel[index] = 0.5/3.14159/pow(sigma,2.0)*pow(2.71828,-(pow(i,2) + pow(j,2))/2/(pow(sigma,2)));  
                sum = sum +  kernel[index];   
        }
    }

    for (int i=0; i<(2*radius + 1)*(2*radius + 1); i++){
        kernel[i] = kernel[i] / sum;
    }
}


__global__ void gradient_rand(float* imgOut_cuda,float* cuda_v1,float* cuda_v2,int w,int h,int nc){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    if (t_numx < w && t_numy < h && t_numz < nc){
        int x_neg = t_numx-1;
        int x_pos = t_numx+1;
        int y_neg = t_numy-1;
        int y_pos = t_numy+1;
        if (x_neg<0) x_neg = 0;
        if (x_pos>=w) x_pos = w-1;
        if (y_neg<0) y_neg = 0;
        if (y_pos>=h) y_pos = h-1;

        cuda_v1[t_numx + w*t_numy + w*h*t_numz] = 1.f/32.f*(    3*imgOut_cuda[x_pos + w*y_pos + w*h*t_numz] 
                                                            +   10*imgOut_cuda[x_pos + w*t_numy + w*h*t_numz] 
                                                            +   3*imgOut_cuda[x_pos + w*y_neg + w*h*t_numz] 
                                                            -   3*imgOut_cuda[x_neg + w*y_pos + w*h*t_numz]
                                                            -   10*imgOut_cuda[x_neg + w*t_numy + w*h*t_numz] 
                                                            -   3*imgOut_cuda[x_neg + w*y_neg + w*h*t_numz]) ;

        cuda_v2[t_numx + w*t_numy + w*h*t_numz] = 1.f/32.f*(    3*imgOut_cuda[x_pos + w*y_pos + w*h*t_numz] 
                                                            +   10*imgOut_cuda[t_numx + w*y_pos + w*h*t_numz] 
                                                            +   3*imgOut_cuda[x_neg + w*y_pos + w*h*t_numz] 
                                                            -   3*imgOut_cuda[x_pos + w*y_neg + w*h*t_numz]
                                                            -   10*imgOut_cuda[t_numx + w*y_neg + w*h*t_numz] 
                                                            -   3*imgOut_cuda[x_neg + w*y_neg + w*h*t_numz]) ;
    }
}

__global__ void cuda_convolution(float *cuda_imgIn, float* imgOut_cuda, float* kernel_cuda, int w, int h, int nc, int radius){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    float output = 0;
    if (t_numx < w && t_numy < h && t_numz < nc){
        int index = t_numx + w*t_numy + w*h*t_numz;
        //imgOut_cuda[index] = 0;
        for (int p = -radius; p <= radius; p++) {
            for (int q =-radius; q <= radius; q++) {
                int temp_j = t_numy + p;
                int temp_i = t_numx + q;
                if (temp_i<0) temp_i = 0;
                if (temp_i>=w) temp_i = w-1;
                if (temp_j<0) temp_j = 0;
                if (temp_j>=h) temp_j = h-1;
                int image_index = temp_i + temp_j*w + t_numz*h*w;
                int kernel_index = q+radius + (2*radius+1)*(p+radius);
                 output += cuda_imgIn[image_index] * kernel_cuda[kernel_index];
            }
        }
    imgOut_cuda[index] = output;
    }    
}

__global__ void M_calculation(float *M_out_cuda, float* cuda_v1, float* cuda_v2, int w, int h, int nc){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    int index = t_numx + w*t_numy + w*h*t_numz;
    if (t_numx < w && t_numy < h && t_numz < nc){
        M_out_cuda[t_numx + w*t_numy + w*h*0] += cuda_v1[index]*cuda_v1[index];
        M_out_cuda[t_numx + w*t_numy + w*h*1] += cuda_v1[index]*cuda_v2[index];
        M_out_cuda[t_numx + w*t_numy + w*h*2] += cuda_v2[index]*cuda_v2[index];
    }

}


__device__ void eigenvalues_eigenvectors (float m11,float m12,float m22, float* lambda1,float* lambda2, float* vector1, float* vector2){
    float T = m11 + m22;
    float D =  m11*m22 - m12*m12;

    *lambda1 = T/2.f + sqrt(pow(T,2.f)/4.f - D);
    *lambda2 = T/2.f - sqrt(pow(T,2.f)/4.f - D);

    if (m12 ==0){
        vector1[0] = 1;
    vector1[1] = 0;
    vector2[0] = 0;
    vector2[1] = 1;
    
    }

    /*else if ((m11-*lambda1)/m12 == m12/(m22-*lambda1)){
        vector1[0] = -m12/(m11-*lambda1);
        vector1[1] = 1;  
    }
    else if ((m11-*lambda2)/m12 == m12/(m22-*lambda2)){
        vector2[0] = -m12/(m11-*lambda2);
        vector2[1] = 1;     
    }*/

}

__global__ void diffusion(float* cuda_v1, float* cuda_v2, float* G_out_cuda, int  w, int h, int nc, float alpha, float C){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    float lambda1;
    float lambda2;
    float vector1[2] = {0,0};
    float vector2[2] = {0,0};
    float m11, m12, m22;
    float mu_1 = alpha;
    float mu_2;
    float g[4];
    int index = t_numx + w*t_numy + w*h*t_numz;
    if (t_numx < w && t_numy < h && t_numz <nc){
       m11 = G_out_cuda[t_numx + w*t_numy + w*h*0];
       m12 = G_out_cuda[t_numx + w*t_numy + w*h*1];
       m22 = G_out_cuda[t_numx + w*t_numy + w*h*2];


        eigenvalues_eigenvectors(m11, m12, m22, &lambda1, &lambda2, vector1, vector2);
        if (lambda1 == lambda2){
            mu_2 = alpha;
        }
        else{
            mu_2 = alpha + (1 - alpha)*exp(-C/pow(lambda1 - lambda2, 2));
        }

        g[0] = mu_1*pow((float)vector1[0] ,(float)2) + mu_2*pow((float)vector2[0] ,(float)2);
        g[1] = mu_1*vector1[0]*vector1[1] + mu_2*vector2[0]*vector2[1]; 
        g[2] = mu_1*vector1[0]*vector1[1] + mu_2*vector2[0]*vector2[1];
        g[3] = mu_1*pow((float)vector1[1] ,(float)2) + mu_2*pow((float)vector2[1], (float)2); 

        cuda_v1[index] = g[0]*cuda_v1[index] + g[1]*cuda_v2[index];
        cuda_v2[index] = g[2]*cuda_v1[index] + g[3]*cuda_v2[index];
    }
}

__global__ void divergence_update(float* cuda_imgIn,float* cuda_div , float* cuda_v1, float* cuda_v2, int w, int h, int nc, float tau ){
    int t_numx = threadIdx.x + blockIdx.x*blockDim.x;
    int t_numy = threadIdx.y + blockIdx.y*blockDim.y;
    int t_numz = threadIdx.z + blockIdx.z*blockDim.z;

    int index = t_numx + w*t_numy + w*h*t_numz;
    if (t_numx > 0 && t_numx < w && t_numy < h && t_numz < nc){
        cuda_div[index] +=  cuda_v1[index] - cuda_v1[index -1];
    }
__syncthreads();
    if (t_numy > 0 && t_numx < w && t_numy < h && t_numz < nc){
        cuda_div[index] += cuda_v2[index] - cuda_v2[index - w];
    }
__syncthreads();
    if (t_numx < w && t_numy < h && t_numz < nc){
        cuda_imgIn[index] += (float)tau*cuda_div[index]; 
    }
}


int main(int argc, char **argv)
{
    hipDeviceSynchronize();  CUDA_CHECK;

#ifdef CAMERA
#else
    // input image
    string image = "";
    float sigma = 0.5;
    float rho = 3;
    int iterations = 10;
    float tau = 0.01;
    float alpha = 0.01;
    float C= 0.000005;

    bool ret = getParam("i", image, argc, argv);
        if (!ret) cerr << "ERROR: no image specified" << endl;

    bool ret2 = getParam("iter", iterations, argc, argv);
        if (!ret2) {cerr << "ERROR: Num of iterations not specified" << endl; return 1;}
        
    bool ret3 = getParam("C", C, argc, argv);
        if (!ret3) {cerr << "ERROR: no C specified" << endl; return 1;}
        
    bool ret4 = getParam("alpha", alpha, argc, argv);
        if (!ret4) {cerr << "ERROR: no alpha specified" << endl; return 1;}
       
    bool ret5 = getParam("sigma", sigma, argc, argv);
        if (!ret5) {cerr << "ERROR: no sigma specified" << endl; return 1;}

    bool ret6 = getParam("rho", rho, argc, argv);
        if (!ret6) {cerr << "ERROR: no rho specified" << endl; return 1;}

    bool ret7 = getParam("tau", tau, argc, argv);
        if (!ret7) {cerr << "ERROR: no tau specified" << endl; return 1;}
        
    if (argc <= 7) { cout << "Usage: " << argv[0] << " -i <image> -iter <iterations> -C <C> -alpha <alpha> -sigma <sigma> -rho <rho> -tau <tau> [-repeats <repeats>] [-gray]" << endl; return 1; }
    #endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
    cv::VideoCapture camera(0);
    if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
    camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
    camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

/// Calculating 1st kernel
    int radius = ceil(3*sigma);
    int total_ele_filter = (int)pow(2*radius + 1, 2);
    float* kernel = new float[total_ele_filter];
    gaussian_kernel(kernel, sigma, radius);
    float *kernel_cuda;
    hipMalloc((void**)&kernel_cuda, total_ele_filter*sizeof(float));
    hipMemcpy(kernel_cuda, kernel, total_ele_filter*sizeof(float), hipMemcpyHostToDevice);

///  calculating different kernel
    int radius_diff = ceil(3*rho);
    int total_ele_filter_diff = (int)pow(2*radius_diff + 1, 2);
    float* kernel_diff = new float[total_ele_filter_diff];
    gaussian_kernel(kernel_diff, rho, radius_diff);
    float *kernel_cuda_diff;
    hipMalloc((void**)&kernel_cuda_diff, total_ele_filter_diff*sizeof(float));
    hipMemcpy(kernel_cuda_diff, kernel_diff, total_ele_filter_diff*sizeof(float), hipMemcpyHostToDevice);	

/// Allocating memory for image data and result in cuda
    int array_size = w*h*nc;
    float *cuda_imgIn, *imgOut_cuda;
    hipMalloc((void**)&cuda_imgIn , array_size*sizeof(float));
    hipMalloc((void**)&imgOut_cuda , array_size*sizeof(float));
    hipMemcpy(cuda_imgIn, imgIn , array_size*sizeof(float) , hipMemcpyHostToDevice); 

/// Defining the grid and block size. 
    dim3 block = dim3(32,32,1);
    int grid_x = ((w + block.x - 1)/block.x);
    int grid_y = ((h + block.y - 1)/block.y);
    int grid_z = ((nc + block.z - 1)/block.z);
    dim3 grid = dim3(grid_x, grid_y, grid_z );
   
/// Allocating memory for forward differencs calculation on cuda 
    float* cuda_v1;
    float* cuda_v2;
    hipMalloc((void**) &cuda_v1, array_size*sizeof(float));
    hipMalloc((void**) &cuda_v2, array_size*sizeof(float));

// Allocating memory for the M and G mmatrix on the cuda
    float *M_out_cuda, *G_out_cuda;
    hipMalloc((void**)&M_out_cuda , array_size*sizeof(float));
    hipMalloc((void**)&G_out_cuda , array_size*sizeof(float));
  
/// Allocating memory for the divergence on cuda
    float *cuda_div;
    hipMalloc((void**) &cuda_div , array_size*sizeof(float));

    for (int iter=0; iter<iterations; iter++){

        hipMemset(imgOut_cuda, 0 , array_size*sizeof(float));   
        cuda_convolution <<<grid, block>>> (cuda_imgIn, imgOut_cuda, kernel_cuda, w, h, nc, radius);

        hipMemset(cuda_v1, 0 , array_size*sizeof(float));
        hipMemset(cuda_v2, 0 , array_size*sizeof(float));

        gradient_rand <<<grid, block>>>(imgOut_cuda, cuda_v1, cuda_v2, w, h, nc );

        hipMemset(M_out_cuda, 0 , array_size*sizeof(float));
        M_calculation <<< grid, block >>> (M_out_cuda, cuda_v1, cuda_v2, w, h, nc );

        hipMemset(G_out_cuda, 0 , array_size*sizeof(float));  
        cuda_convolution <<< grid, block >>> (M_out_cuda, G_out_cuda, kernel_cuda_diff, w, h, nc, radius_diff);

        diffusion <<< grid, block >>> ( cuda_v1, cuda_v2, G_out_cuda, w, h, nc, alpha, C);
        //hipMemcpy(imgOut, cuda_v1 , w*h*nc*sizeof(float) , hipMemcpyDeviceToHost);

        hipMemset(cuda_div, 0, array_size*sizeof(float));
        divergence_update <<< grid, block >>> (cuda_imgIn, cuda_div , cuda_v1, cuda_v2, w, h, nc, tau );
}
    
    hipMemcpy(imgOut, cuda_imgIn , array_size*sizeof(float) , hipMemcpyDeviceToHost);

    convert_layered_to_mat(mOut,imgOut);
    showImage("OUTPUT", mOut, 0, 0);
    showImage("Input", mIn, 100, 100); 
#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] kernel;
    delete[] kernel_diff;

    hipFree(cuda_imgIn);
    hipFree(cuda_v2);
    hipFree(cuda_v1);
    hipFree(cuda_div);
    hipFree(G_out_cuda);
    hipFree(M_out_cuda);
    hipFree(kernel_cuda);
    hipFree(kernel_cuda_diff);
    hipFree(imgOut_cuda);

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



